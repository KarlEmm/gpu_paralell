
#include <hip/hip_runtime.h>
#include <iostream>

#define MATRIX_SIZE 4

#define CUDAMALLOC_ERROR(_err) \
do {                           \
    if (_err != hipSuccess) { \
        printf("%s failed in file %s at line #%d\n", hipGetErrorString(_err),__FILE__,__LINE__); \
        exit(EXIT_FAILURE);   \
        }                      \
    } while(0)

void fillMatrix(float *matrix, float value)
{
    for (int i = 0; i < MATRIX_SIZE*MATRIX_SIZE; ++i)
    {
        matrix[i] = value;
    }
}

__global__
void matrixAddKernel(float *A, float *B, float *C, size_t n)
{
    unsigned int Col = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int Row = blockIdx.y * blockDim.y + threadIdx.y;

    if (Row < n && Col < n)
    {
        // 1D coordinates
        int coord = Row * n + Col;
        C[coord] = A[coord] + B[coord];
    }
}

void addMatrices()
{
    hipDeviceProp_t dev_prop;
    hipGetDeviceProperties(&dev_prop, 0);

    // For this machine, sqrt(1024) = 32. Therefore, we have a 32-square matrix per block.
    dim3 dimBlock(sqrt(dev_prop.maxThreadsPerBlock),sqrt(dev_prop.maxThreadsPerBlock),1);
    dim3 dimGrid(1,1,1);

    int matrixSize = MATRIX_SIZE * MATRIX_SIZE;
    int matrixByteSize = matrixSize * sizeof (float);

    float *h_A;
    float *h_B;
    float *h_C;

    h_A = (float *) malloc (matrixByteSize);
    h_B = (float *) malloc (matrixByteSize);
    h_C = (float *) malloc(matrixByteSize);

    fillMatrix(h_A, 1);
    fillMatrix(h_B, 2);
    fillMatrix(h_C, 0);

    float *d_A;
    hipError_t err = hipMalloc((void**)&d_A, matrixByteSize);
    CUDAMALLOC_ERROR(err);
    float *d_B;
    err = hipMalloc((void**)&d_B, matrixByteSize);
    CUDAMALLOC_ERROR(err);
    float *d_C;
    err = hipMalloc((void**)&d_C, matrixByteSize);
    CUDAMALLOC_ERROR(err);

    hipMemcpy(d_A, h_A, matrixByteSize, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, matrixByteSize, hipMemcpyHostToDevice);

    matrixAddKernel<<<dimGrid, dimBlock>>>(d_A, d_B, d_C, MATRIX_SIZE);

    hipMemcpy(h_C, d_C, matrixByteSize, hipMemcpyDeviceToHost);

    for (int i = 0 ; i < 16; ++i) {
        h_A[i] = 0;
    }

    hipMemcpy(h_A, d_A, matrixByteSize, hipMemcpyDeviceToHost);

    for (int i = 0; i < 15; ++i) {
        std::cout << h_C[i] << std::endl;
    }


    hipFree(d_A); hipFree(d_B); hipFree(d_C);
}


int main() {
    addMatrices();
    return 0;
}
